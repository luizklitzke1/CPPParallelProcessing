#include "hip/hip_runtime.h"
﻿
#include <random>
#include <stdio.h>
#include <cassert>

//Libs da NVidia para uso de CUDA Cores
#include "hip/hip_runtime.h"
#include ""

#define BLOCKS 1

//Limite de threas por bloco = 1024
#define THREADS_PER_BLOCK 1024

__global__ void addKernel(const int* vectorA, const int* vectorB, int* sumVector)
{
    const int idxThread = threadIdx.x;
    sumVector[idxThread] = vectorA[idxThread] + vectorB[idxThread];
}

hipError_t addWithCuda(const int* vectorA, const int* vectorB, int* sumVector)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    hipError_t cudaStatus = hipError_t::hipSuccess;

    // Definir qual device vai ser utilizado
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        printf("Erro ao buscar um hipSetDevice. Verifique se sua GPU é compatível");
        goto FreeCuda;
    }
    
    { // Alocação de buffer de GPU para os vetores
        cudaStatus = hipMalloc((void**)&dev_a, THREADS_PER_BLOCK * sizeof(int));
        if (cudaStatus != hipSuccess) 
        {
            printf("Erroi ao alocar memória do vetor A - hipMalloc()");
            goto FreeCuda;
        }

        cudaStatus = hipMalloc((void**)&dev_b, THREADS_PER_BLOCK * sizeof(int));
        if (cudaStatus != hipSuccess) 
        {
            printf("Erro ao alocar memória do vetor B - hipMalloc()");
            goto FreeCuda;
        }

        cudaStatus = hipMalloc((void**)&dev_c, THREADS_PER_BLOCK * sizeof(int));
        if (cudaStatus != hipSuccess)
        {
            printf("Erro ao alocar memória do vetor de Soma - hipMalloc()");
            goto FreeCuda;
        }
    }

    { // Copiar memória dos vetores para o Buffer da GPU
        cudaStatus = hipMemcpy(dev_a, vectorA, THREADS_PER_BLOCK * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) 
        {
            printf("Erro ao copiar os valores do vetor A - hipMemcpy()");
            goto FreeCuda;
        }

        cudaStatus = hipMemcpy(dev_b, vectorB, THREADS_PER_BLOCK * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) 
        {
            printf("Erro ao copiar os valores do vetor B - hipMemcpy()");
            goto FreeCuda;
        }

    }
    
    //Cahmada do Kernel poara processamento paralelo, com um único bloco contendo uma threada para cada index do vetor
    addKernel << <BLOCKS, THREADS_PER_BLOCK >> > (dev_a, dev_b, dev_c);

    //Validar erros na chamada de Kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        printf("Erro ao executar addKernel() - Cod %d - %s\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto FreeCuda;
    }

    //Validar sincronização dos devices após executar chamada de Kernel
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        printf("Erro ao executar hipDeviceSynchronize %d  - Cod %d - %s \n", cudaStatus, hipGetErrorString(cudaStatus));
        goto FreeCuda;
    }

    //Copiar dados do buffer de memória da GPU - managed - de volta para memória local do host
    cudaStatus = hipMemcpy(sumVector, dev_c, THREADS_PER_BLOCK * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
    {
        printf("Erro ao copiar memória do buffer da GPU  - hipMemcpy()");
        goto FreeCuda;
    }

FreeCuda:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

int main()
{
    int vectorA  [THREADS_PER_BLOCK] = { 0 };
    int vectorB  [THREADS_PER_BLOCK] = { 0 };
    int sumvector[THREADS_PER_BLOCK] = { 0 };

    {// Popular vetores com inteiros aleatórios - https://stackoverflow.com/questions/13445688/how-to-generate-a-random-number-in-c
        std::random_device device;
        std::mt19937 rng(device());

        std::uniform_int_distribution<std::mt19937::result_type> getRandInt(0, (INT_MAX / 2) - 1);

        for (int i = 0; i < THREADS_PER_BLOCK; ++i)
        {
            vectorA[i] = getRandInt(rng);
            vectorB[i] = getRandInt(rng);
        }
    }

    hipError_t cudaStatus = addWithCuda(vectorA, vectorB, sumvector);
    if (cudaStatus != hipSuccess)
    {
        printf("Erro ao processar soma em CUDA");
        return 1;
    }

    {//Validar somas
        for (int i = 0; i < THREADS_PER_BLOCK; ++i)
        {
            const int valueA   = vectorA  [i];
            const int valueB   = vectorB  [i];
            const int sumValue = sumvector[i];

            assert(sumValue == valueA + valueB);

            if (sumValue != valueA + valueB)
            {
                printf("[%d][ERRO DE SOMA][Diferença encontrada na soma! - %d + %d != %d\n", i, valueA, valueB, sumValue);
                return 1;
            }

            printf("[%d]%d + %d = %d\n", i, valueA, valueB, sumValue);
        }
    }

    {//Limpar devices para evitar erros de profiling
        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess)
        {
            printf("Erro ao executar hipDeviceReset()");
            return 1;
        }
    }

    return 0;
}
