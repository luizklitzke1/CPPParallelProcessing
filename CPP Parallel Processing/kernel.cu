#include "hip/hip_runtime.h"
﻿
#include <random>
#include <stdio.h>
#include <cassert>

//Libs da NVidia para uso de CUDA Cores
#include "hip/hip_runtime.h"
#include ""

#define ARRAY_SIZE 25

__global__ void addKernel(const int* vectorA, const int* vectorB, int* sumVector)
{
    const int idxThread = threadIdx.x;
    sumVector[idxThread] = vectorA[idxThread] + vectorB[idxThread];
}

hipError_t addWithCuda(const int* vectorA, const int* vectorB, int* sumVector)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    hipError_t cudaStatus = hipError_t::hipSuccess;

    // Definir qual device vai ser utilizado
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        printf("Erro ao buscar um hipSetDevice. Verifique se sua GPU é compatível");
        goto FreeCuda;
    }
    
    { // Alocação de buffer de GPU para os vetores
        cudaStatus = hipMalloc((void**)&dev_a, ARRAY_SIZE * sizeof(int));
        if (cudaStatus != hipSuccess) 
        {
            printf("Erroi ao alocar memória do vetor A - hipMalloc()");
            goto FreeCuda;
        }

        cudaStatus = hipMalloc((void**)&dev_b, ARRAY_SIZE * sizeof(int));
        if (cudaStatus != hipSuccess) 
        {
            printf("Erro ao alocar memória do vetor B - hipMalloc()");
            goto FreeCuda;
        }

        cudaStatus = hipMalloc((void**)&dev_c, ARRAY_SIZE * sizeof(int));
        if (cudaStatus != hipSuccess)
        {
            printf("Erro ao alocar memória do vetor de Soma - hipMalloc()");
            goto FreeCuda;
        }
    }

    { // Copiar memória dos vetores para o Buffer da GPU
        cudaStatus = hipMemcpy(dev_a, vectorA, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) 
        {
            printf("Erro ao copiar os valores do vetor A - hipMemcpy()");
            goto FreeCuda;
        }

        cudaStatus = hipMemcpy(dev_b, vectorB, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) 
        {
            printf("Erro ao copiar os valores do vetor B - hipMemcpy()");
            goto FreeCuda;
        }

    }
    
    //Cahmada do Kernel poara processamento paralelo, com um único bloco contendo uma threada para cada index do vetor
    addKernel << <1, ARRAY_SIZE >> > (dev_a, dev_b, dev_c);

    //Validar erros na chamada de Kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        printf("Erro ao executar addKernel() - Cod %d - %s\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto FreeCuda;
    }

    //Validar sincronização dos devices após executar chamada de Kernel
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        printf("Erro ao executar hipDeviceSynchronize %d  - Cod %d - %s \n", cudaStatus, hipGetErrorString(cudaStatus));
        goto FreeCuda;
    }

    //Copiar dados do buffer de memória da GPU - managed - de volta para memória local do host
    cudaStatus = hipMemcpy(sumVector, dev_c, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
    {
        printf("Erro ao copiar memória do buffer da GPU  - hipMemcpy()");
        goto FreeCuda;
    }

FreeCuda:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

int main()
{
    int vectorA  [ARRAY_SIZE] = { 0 };
    int vectorB  [ARRAY_SIZE] = { 0 };
    int sumvector[ARRAY_SIZE] = { 0 };

    {// Popular vetores com inteiros aleatórios - https://stackoverflow.com/questions/13445688/how-to-generate-a-random-number-in-c
        std::random_device device;
        std::mt19937 rng(device());

        std::uniform_int_distribution<std::mt19937::result_type> getRandInt(0, (INT_MAX / 2) - 1);

        for (int i = 0; i < ARRAY_SIZE; ++i)
        {
            vectorA[i] = getRandInt(rng);
            vectorB[i] = getRandInt(rng);
        }
    }

    hipError_t cudaStatus = addWithCuda(vectorA, vectorB, sumvector);
    if (cudaStatus != hipSuccess)
    {
        printf("Erro ao processar soma em CUDA");
        return 1;
    }

    {//Validar somas
        for (int i = 0; i < ARRAY_SIZE; ++i)
        {
            const int valueA   = vectorA  [i];
            const int valueB   = vectorB  [i];
            const int sumValue = sumvector[i];

            assert(sumValue == valueA + valueB);

            if (sumValue != valueA + valueB)
            {
                printf("[%d][ERRO DE SOMA][Diferença encontrada na soma! - %d + %d != %d\n", i, valueA, valueB, sumValue);
                return 1;
            }

            printf("[%d]%d + %d = %d\n", i, valueA, valueB, sumValue);
        }
    }

    {//Limpar devices para evitar erros de profiling
        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess)
        {
            printf("Erro ao executar hipDeviceReset()");
            return 1;
        }
    }

    return 0;
}
