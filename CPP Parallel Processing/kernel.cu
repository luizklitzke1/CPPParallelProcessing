#include "hip/hip_runtime.h"
﻿
#include <random>
#include <stdio.h>
#include <cassert>
#include <chrono>
#include <thread>
#include <ppl.h>
#include <iostream>
#include <string>
#include <format>

#include "windows.h"

//Libs da NVidia para uso de CUDA Cores
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

using msTime = std::chrono::duration<double, std::milli>;
using UINT = unsigned int;

FILE* fp;

int getSPcores(hipDeviceProp_t devProp)
{  
    int cores = 0;
    int mp = devProp.multiProcessorCount;

    switch (devProp.major)
    {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else fprintf(fp,"Unknown device type\n");
            break;
        case 7: // Volta and Turing
            if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
            else fprintf(fp,"Unknown device type\n");
            break;
        case 8: // Ampere
            if (devProp.minor == 0) cores = mp * 64;
            else if (devProp.minor == 6) cores = mp * 128;
            else if (devProp.minor == 9) cores = mp * 128; // ada lovelace
            else fprintf(fp,"Unknown device type\n");
            break;
        case 9: // Hopper
            if (devProp.minor == 0) cores = mp * 128;
            else fprintf(fp,"Unknown device type\n");
            break;
        default:
            fprintf(fp,"Unknown device type\n");
            break;
    }

    return cores;
}

__global__ void KernelMatrixVectorProduct(float* A, float* v1, float* v2, UINT uiMatrixSize)
{
    const int iMatrixRow = blockIdx.x * blockDim.x + threadIdx.x;
    const int iMatrixCol = blockIdx.y * blockDim.y + threadIdx.y;

    if (iMatrixCol == 0 && iMatrixRow < uiMatrixSize)
    {
        float fSum = 0.0f;

        for (int i = 0; i < uiMatrixSize; ++i)
        {
            fSum += A[iMatrixRow * uiMatrixSize + i] * v1[i];
        }

        v2[iMatrixRow] = fSum;
    }
}

hipError_t CUDAMatrixVectorProduct(float* A, float* v1, float* v2, UINT uiMatrixSize, msTime& processingTime)
{
    fprintf(fp,"\n\n[PRODUTO DE VETOR X MATRIZ - CUDA CORES - INÍCIO]\n");

    float* A_GPU ;
    float* v1_GPU;
    float* v2_GPU;

    dim3 block_shape = dim3(32, 32);
    dim3 grid_shape  = dim3(max(1.0, std::ceil((float)uiMatrixSize / (float)block_shape.x)),
                            max(1.0, std::ceil((float)uiMatrixSize / (float)block_shape.y)));

    hipError_t cudaStatus = hipError_t::hipSuccess;

    // Definir qual device vai ser utilizado
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(fp,"Erro ao buscar um hipSetDevice. Verifique se sua GPU é compatível");
        goto FreeCuda;
    }

    // Log de specs do device
    {
        int deviceID;
        hipDeviceProp_t devProps;

        cudaStatus = hipGetDevice(&deviceID);
        if (cudaStatus != hipSuccess) {
            fprintf(fp,"Erro ao pegar ID do device - hipGetDevice() - Cod %d - %s \n", cudaStatus, hipGetErrorString(cudaStatus));
            goto FreeCuda;
        }

        hipGetDeviceProperties(&devProps, deviceID);
        const int iCUDACores = getSPcores(devProps);

        fprintf(fp,"Device \"%s\" selecionado.\n", devProps.name);
        fprintf(fp,"CUDA cores: %d\t| Multiprocessadores: %d\t| Warp size: %d\n", iCUDACores, devProps.multiProcessorCount, devProps.warpSize);
        fprintf(fp,"Max Blocks Per MultiProcessor: %d\t| Max Threads per block: %d\n", devProps.maxBlocksPerMultiProcessor, devProps.maxThreadsPerBlock);
        fprintf(fp,"Block Shape: %d - %d - %d\n", block_shape.x, block_shape.y, block_shape.z);
        fprintf(fp,"Grid  Shape: %d - %d - %d\n", grid_shape .x, grid_shape .y, grid_shape .z);
    }

    auto clockInicioCuda = std::chrono::high_resolution_clock::now();
   
    // Alocação de buffer de GPU para os vetores
    {
        cudaStatus = hipMalloc((void**)&A_GPU, uiMatrixSize * uiMatrixSize * sizeof(float));
        if (cudaStatus != hipSuccess) 
        {
            fprintf(fp,"Erroi ao alocar memória da matriz A - hipMalloc()");
            goto FreeCuda;
        }

        cudaStatus = hipMalloc((void**)&v1_GPU, uiMatrixSize * sizeof(float));
        if (cudaStatus != hipSuccess) 
        {
            fprintf(fp,"Erro ao alocar memória do vetor 1 - hipMalloc()");
            goto FreeCuda;
        }

        cudaStatus = hipMalloc((void**)&v2_GPU, uiMatrixSize * sizeof(float));
        if (cudaStatus != hipSuccess)
        {
            fprintf(fp,"Erro ao alocar memória do vetor 2 - hipMalloc()");
            goto FreeCuda;
        }
    }

    // Copiar memória dos vetores para o Buffer da GPU
    { 
        cudaStatus = hipMemcpy(A_GPU, A, uiMatrixSize * uiMatrixSize * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) 
        {
            fprintf(fp,"Erro ao copiar os valores da matriz A - hipMemcpy()");
            goto FreeCuda;
        }

        cudaStatus = hipMemcpy(v1_GPU, v1, uiMatrixSize * sizeof(float), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) 
        {
            fprintf(fp,"Erro ao copiar os valores do vetor 1 - hipMemcpy()");
            goto FreeCuda;
        }

    }

    KernelMatrixVectorProduct << <grid_shape, block_shape >> > (A_GPU, v1_GPU, v2_GPU, uiMatrixSize);

    //Validar erros na chamada de Kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(fp,"Erro ao executar chamada do kernel - Cod %d - %s\n", cudaStatus, hipGetErrorString(cudaStatus));
        goto FreeCuda;
    }

    //Validar sincronização dos devices após executar chamada de Kernel
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        fprintf(fp,"Erro ao executar hipDeviceSynchronize %d - Cod %d - %s \n", cudaStatus, hipGetErrorString(cudaStatus));
        goto FreeCuda;
    }

    //Copiar dados do buffer de memória da GPU - managed - de volta para memória local do host
    cudaStatus = hipMemcpy(v2, v2_GPU, uiMatrixSize * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(fp,"Erro ao copiar memória do buffer da GPU  - hipMemcpy()");
        goto FreeCuda;
    }

FreeCuda:
    hipFree(A_GPU );
    hipFree(v1_GPU);
    hipFree(v2_GPU);

    auto clockFimCuda = std::chrono::high_resolution_clock::now();
    processingTime = clockFimCuda - clockInicioCuda;
    fprintf(fp,"Tempo total de processamento com CUDA cores: %fms\n", processingTime.count());

    fprintf(fp,"[CUDA CORES - FIM]\n");

    return cudaStatus;
}

void linearMatrixVectorProduct(float *A, float* v1, float* v2, UINT uiMatrixSize, msTime& processingTime)
{
    fprintf(fp,"\n\n[PRODUTO DE VETOR X MATRIZ - PROCESSAMENTO LINEAR - INÍCIO]\n");

    auto clockInicioLinear = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < uiMatrixSize; ++i)
    {
        float fSum = 0.0f;

        for (int j = 0; j < uiMatrixSize; ++j)
        {
            fSum += A[i * uiMatrixSize + j] * v1[j];
        }

        v2[i] = fSum;
    }

    auto clockFimLinear = std::chrono::high_resolution_clock::now();

    processingTime = clockFimLinear - clockInicioLinear;

    fprintf(fp,"Tempo total de processamento linear: %fms\n", processingTime.count());
    fprintf(fp,"[PRODUTO DE VETOR X MATRIZ - PROCESSAMENTO LINEAR - FIM]\n");
}

void CPUConcurrencyMatrixVectorProduct(float* A, float* v1, float* v2, UINT uiMatrixSize, msTime& processingTime)
{
    fprintf(fp,"\n\n[PRODUTO DE VETOR X MATRIZ - PROCESSAMENTO CONCORRENTE EM CPU - INÍCIO]\n");

    const UINT uiSupportedThreads = std::thread::hardware_concurrency();

    fprintf(fp,"Quantidade de threads suportadas pela CPU: %hd\n", uiSupportedThreads);

    auto clockInicio = std::chrono::high_resolution_clock::now();

    Concurrency::parallel_for<int>(0, uiMatrixSize, [&](int i)
    {
        float fSum = 0.0f;

        for (int j = 0; j < uiMatrixSize; ++j)
        {
            fSum += A[i * uiMatrixSize + j] * v1[j];
        }

        v2[i] = fSum;
    });

    auto clockFim = std::chrono::high_resolution_clock::now();

    processingTime = clockFim - clockInicio;

    fprintf(fp,"Tempo total de processamento concorrente em CPU: %fms\n", processingTime.count());
    fprintf(fp,"[PRODUTO DE VETOR X MATRIZ - PROCESSAMENTO CONCORRENTE EM CPU - FIM]\n");
}

int main(int argc, char **argv)
{
    SetConsoleCP      (1252);
    SetConsoleOutputCP(1252);

    UINT uiMatrixSizeCFG = 0;

    {
        const std::string sTitulo = "[Benchmark de processamento paralelo]";
        const std::string sOperacao = "Multiplicação de matrix NxN por vetor N - (N Sendo um número inteiro > 0)";

        printf("%s\,", sTitulo.c_str());
        printf("\n[Configurações]\nOperação: %s\n", sOperacao.c_str());

        while (std::cout << "Informe o valor de N: " && !(std::cin >> uiMatrixSizeCFG)) {
            std::cin.clear(); //clear bad input flag
            std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n'); //discard input
            std::cout << "Valor inválido\n";
        }

        fp = fopen("result.txt", "a");
        fprintf(fp, "%s\n", sTitulo.c_str());
        fprintf(fp, "\n[Configurações]\nOperação: %s\n", sOperacao.c_str());
    }

    fprintf(fp, "Valor de N: %d\n", uiMatrixSizeCFG);

    float* A ; // Matriz N * N
    float* v1; // Vetor para mult

    float* v2Linear;
    float* v2CPU   ;
    float* v2CUDA  ;

    A  = (float*)malloc(uiMatrixSizeCFG * uiMatrixSizeCFG * sizeof(float));
    v1 = (float*)malloc(uiMatrixSizeCFG  * sizeof(float));

    v2Linear = (float*)malloc(uiMatrixSizeCFG  * sizeof(float));
    v2CPU    = (float*)malloc(uiMatrixSizeCFG  * sizeof(float));
    v2CUDA   = (float*)malloc(uiMatrixSizeCFG  * sizeof(float));

    // Popular vetores com valores reais aleatórios
    {
        std::random_device device; //Gerar seed
        std::mt19937 rng(device());

        std::uniform_real_distribution<> getRandReal(0.1, 999.9);

        for (int i = 0; i < uiMatrixSizeCFG; ++i)
        {
            //A é uma matrix N * N, porém representada linearmente para facilitar blocos de CUDA posteriormente
            for (int j = 0; j < uiMatrixSizeCFG; ++j)
            {
                A[i * uiMatrixSizeCFG + j] = getRandReal(rng);
            }

            v1[i] = getRandReal(rng);
        }
    }

    //Processamento Linear
    msTime linearProcessingTime;
    linearMatrixVectorProduct(A, v1, v2Linear, uiMatrixSizeCFG, linearProcessingTime);

    //Processamento com concorrencia em CPU
    msTime CPUProcessingTime;
    CPUConcurrencyMatrixVectorProduct(A, v1, v2Linear, uiMatrixSizeCFG, CPUProcessingTime);

    msTime CUDAProcessingTime;
    //Processamento paraleo com CUDA cores
    {
        hipError_t cudaStatus = CUDAMatrixVectorProduct(A, v1, v2CUDA, uiMatrixSizeCFG, CUDAProcessingTime);
        if (cudaStatus != hipSuccess)
        {
            fprintf(fp,"Erro ao processar soma em CUDA");
            return 1;
        }

        // Limpar devices para evitar erros de profiling
        {
            cudaStatus = hipDeviceReset();
            if (cudaStatus != hipSuccess)
            {
                fprintf(fp,"Erro ao executar hipDeviceReset()");
                return 1;
            }
        }
    }

    //Liberar valores dos ponteiros de matrizes
    {
        free(A       );
        free(v1      );
        free(v2Linear);
        free(v2CPU   );
        free(v2CUDA  );
    }

    fprintf(fp,"\n\n[DIF] Diferença entre processamento linear e paralelizado com CPU threads/cores = %fms\n", linearProcessingTime.count() - CPUProcessingTime .count());
    fprintf(fp,"[DIF] Diferença entre processamento linear e paralelizado com CUDA cores        = %fms\n\n\n"       , linearProcessingTime.count() - CUDAProcessingTime.count());

    fclose(fp);

    return 0;
}
